#include "hip/hip_runtime.h"
#include "Kernels.cuh"

void AllocGpuMemory(size_t imageSizeBytes)
{
	gpuErrchk(hipMallocManaged(&inputImageDev, imageSizeBytes));
	gpuErrchk(hipMallocManaged(&tempImageDev, imageSizeBytes));
	gpuErrchk(hipMallocManaged(&outputImageDev, imageSizeBytes));
}

void FreeGpuMemory()
{
	if(inputImageDev != nullptr)
	      gpuErrchk(hipFree(inputImageDev));

	if(tempImageDev != nullptr)
	      gpuErrchk(hipFree(tempImageDev));

	if(outputImageDev != nullptr)
	      gpuErrchk(hipFree(outputImageDev));
}

__device__ void FixIndexes(int &pixelX,  int &pixelY, size_t hightImg, size_t widthImg)
{
	if (pixelX < 0)
		pixelX = 0;

	if (pixelY < 0)
		pixelY = 0;

	if (pixelX >= hightImg)
		pixelX = hightImg - 1;

	if (pixelY >= widthImg)
		pixelY = widthImg - 1;
}

__global__ void ErosionDev(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize, PixelsPtrType outputImage)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int mainPixelX = threadIndex / widthImg;
	int mainPixelY = threadIndex % widthImg;

	if (threadIndex >= widthImg * hightImg)
	{
		return;
	}

	int halfSide = (int)((maskSize - 1) / 2);
	bool foundZero = false;

	for (int row = -halfSide; row <= halfSide && !foundZero; ++row)
	{
		for (int col = -halfSide; col <= halfSide && !foundZero; ++col)
		{
			int currentPixelX = mainPixelX + row;
			int currentPixelY = mainPixelY + col;

			FixIndexes(currentPixelX, currentPixelY, hightImg, widthImg);

			if(inputImage[currentPixelX * widthImg + currentPixelY] == 0)
			{
			  foundZero = true;
			  break;
			}
		}
	}

	if(foundZero)
	{
	  outputImage[threadIndex] = 0;
	}
	else
	{
	  outputImage[threadIndex] = 1;
	}
}

__global__ void DilationDev(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize, PixelsPtrType outputImage)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int mainPixelX = threadIndex / widthImg;
	int mainPixelY = threadIndex % widthImg;

	if (threadIndex >= widthImg * hightImg)
	{
		return;
	}

	int halfSide = (int)((maskSize - 1) / 2);
	bool foundOne = false;

	for (int row = -halfSide; row <= halfSide && !foundOne; ++row)
	{
		for (int col = -halfSide; col <= halfSide && !foundOne; ++col)
		{
			int currentPixelX = mainPixelX + row;
			int currentPixelY = mainPixelY + col;

			FixIndexes(currentPixelX, currentPixelY, hightImg, widthImg);

			if(inputImage[currentPixelX * widthImg + currentPixelY] > 0)
			{
			  foundOne = true;
			  break;
			}
		}
	}

	if(foundOne)
	{
	  outputImage[threadIndex] = 1;
	}
	else
	{
	  outputImage[threadIndex] = 0;
	}
}

PixelsPtrType Erosion(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize)
{
    auto imageSizeBytes = sizeof(PixelType) * widthImg * hightImg;
    auto anglesTableSizeBytes = sizeof(AngleType) * widthImg * hightImg;
    auto blocksNumber = (widthImg * hightImg) / THREAD_NUMBER;
    auto threadsNumber = THREAD_NUMBER;

    gpuErrchk(hipMemcpy(inputImageDev, inputImage, imageSizeBytes, hipMemcpyHostToDevice));
    ErosionDev << <blocksNumber, threadsNumber >> > (inputImageDev, widthImg, hightImg, maskSize, outputImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    auto outputImage = (PixelsPtrType)malloc(sizeof(PixelType) * widthImg * hightImg);

    gpuErrchk(hipMemcpy(outputImage, outputImageDev, imageSizeBytes, hipMemcpyDeviceToHost));

	return outputImage;
}

PixelsPtrType Dilation(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize)
{
    auto imageSizeBytes = sizeof(PixelType) * widthImg * hightImg;
    auto anglesTableSizeBytes = sizeof(AngleType) * widthImg * hightImg;
    auto blocksNumber = (widthImg * hightImg) / THREAD_NUMBER;
    auto threadsNumber = THREAD_NUMBER;

    gpuErrchk(hipMemcpy(inputImageDev, inputImage, imageSizeBytes, hipMemcpyHostToDevice));
    DilationDev << <blocksNumber, threadsNumber >> > (inputImageDev, widthImg, hightImg, maskSize, outputImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    auto outputImage = (PixelsPtrType)malloc(sizeof(PixelType) * widthImg * hightImg);

    gpuErrchk(hipMemcpy(outputImage, outputImageDev, imageSizeBytes, hipMemcpyDeviceToHost));

	return outputImage;
}

PixelsPtrType Opening(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize)
{
    auto imageSizeBytes = sizeof(PixelType) * widthImg * hightImg;
    auto anglesTableSizeBytes = sizeof(AngleType) * widthImg * hightImg;
    auto blocksNumber = (widthImg * hightImg) / THREAD_NUMBER;
    auto threadsNumber = THREAD_NUMBER;

    gpuErrchk(hipMemcpy(inputImageDev, inputImage, imageSizeBytes, hipMemcpyHostToDevice));
    ErosionDev << <blocksNumber, threadsNumber >> > (inputImageDev, widthImg, hightImg, maskSize, tempImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    DilationDev << <blocksNumber, threadsNumber >> > (tempImageDev, widthImg, hightImg, maskSize, outputImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    auto outputImage = (PixelsPtrType)malloc(sizeof(PixelType) * widthImg * hightImg);

    gpuErrchk(hipMemcpy(outputImage, outputImageDev, imageSizeBytes, hipMemcpyDeviceToHost));

	return outputImage;
}

PixelsPtrType Closing(const PixelsPtrType inputImage, const size_t widthImg, const size_t hightImg, const size_t maskSize)
{
    auto imageSizeBytes = sizeof(PixelType) * widthImg * hightImg;
    auto anglesTableSizeBytes = sizeof(AngleType) * widthImg * hightImg;
    auto blocksNumber = (widthImg * hightImg) / THREAD_NUMBER;
	auto threadsNumber = THREAD_NUMBER;

    gpuErrchk(hipMemcpy(inputImageDev, inputImage, imageSizeBytes, hipMemcpyHostToDevice));
    DilationDev << <blocksNumber, threadsNumber >> > (inputImageDev, widthImg, hightImg, maskSize, tempImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    ErosionDev << <blocksNumber, threadsNumber >> > (tempImageDev, widthImg, hightImg, maskSize, outputImageDev);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    auto outputImage = (PixelsPtrType)malloc(sizeof(PixelType) * widthImg * hightImg);

    gpuErrchk(hipMemcpy(outputImage, outputImageDev, imageSizeBytes, hipMemcpyDeviceToHost));

	return outputImage;
}
