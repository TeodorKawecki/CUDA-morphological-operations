#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <memory>
#include "IImgReader.h"
#include "OpenCVImageReader.h"
#include "PbmImageWriter.h"
#include "argagg.hpp"
#include "IOHelpers.h"
#include "IMorphologicalOperation.h"
#include "MorphologicalOperationsTypes.h"
#include "IMorphologicalOperationsFactory.h"
#include "MorphologicalOperationsCudaFactory.h"
#include "MorphologicalOperationsSynchFactory.h"
#include "ArgParserWrapper.h"
#include "ArgParserException.h"
#include "DummyImage.h"
#include "ImageIOException.h"

int main(int argc, char **argv)
{
	ArgParserWrapper argParserWrapper;
	StartingParameters startingParameters;

	try
	{
		startingParameters = argParserWrapper.Parse(argc, argv);
	}
	catch (const ArgParserException &ex)
	{
		std::cerr << ex.what();
		return EXIT_FAILURE;
	}

	try
	{
		auto imageReader = std::make_unique<OpenCVImageReader>(startingParameters.threshold);
		auto imageWriter = std::make_unique<PbmImageWriter>();

		auto inputImage = imageReader->ReadImage(IOHelpers::GetExePath() + startingParameters.inputRelativeFilePath);

		std::unique_ptr<IMorphologicalOperationsFactory> morphologicalOperationsFactory = nullptr;

		if (startingParameters.cudaMode)
		{
			morphologicalOperationsFactory = std::make_unique<MorphologicalOperationsCudaFactory>(inputImage->GetWidth(), inputImage->GetHight(), startingParameters.maskSize);
		}
		else
		{
			morphologicalOperationsFactory = std::make_unique<MorphologicalOperationsSynchFactory>(startingParameters.maskSize);
		}

		auto morphologicalOperation = morphologicalOperationsFactory->Create(startingParameters.operationType);

		std::shared_ptr<IImage> outputImage = std::make_shared<DummyImage>();

		for (int i = 0; i < startingParameters.repeatOperation; ++i)
		{
			auto begin = std::chrono::steady_clock::now();
			outputImage = morphologicalOperation->Process(inputImage);
			auto end = std::chrono::steady_clock::now();

			std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << std::endl;
		}

		if (!startingParameters.surpressWritingImage)
		{
			imageWriter->WriteImage(IOHelpers::GetExePath() + startingParameters.outputRelativeFilePath, outputImage);
		}
	}
	catch (const ImageIOException &e)
	{
		std::cerr << e.what();
		return EXIT_FAILURE;
	}
	catch (const std::exception &e)
	{
		std::cerr << e.what();
		return EXIT_FAILURE;
	}
	
    return 0;
}
